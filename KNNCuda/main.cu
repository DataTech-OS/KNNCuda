#include "hip/hip_runtime.h"
#include <Windows.h>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

using namespace std;

void KNearestNeighborsCPU(float3 *dataArray, int *result, int cnt);
void KNearestNeighborsCPUTreeSearch(float3 *dataArray, int *result, int cnt);
/*__global__ void KNearestNeighborsGPU(float3 *dataArray, int *result, int cnt);
__global__ void KNearestNeighborsGPUTreeSearch(float3 *dataArray, int *result, int cnt);
*/
int cnt = 10000;

int main(int argc, char **argv)
{
	clock_t init, end;

	// generate the data
	srand(time(NULL));

	int timt = 0;
	float3 *dataArray = new float3[cnt];
	int *result = new int[cnt];

	for (int i = 0; i < cnt; i++)
	{
		dataArray[i].x = (rand() / 10000) - 5000;
		dataArray[i].y = (rand() / 10000) - 5000;
		dataArray[i].z = (rand() / 10000) - 5000;
	}
	// first check the speed of the algorithm takes on the cpu
	for (int i = 0; i < 50; i++)
	{
		init = clock();
		KNearestNeighborsCPU(dataArray, result, cnt);
		end = clock();
		timt += end - init;
	}
	cout << "[+] The non optimived version of the algorithm on the CPU takes " << timt / 50 << " milliseconds" << endl;
	timt = 0;

	for (int i = 0; i < 10; i++)
		cout << i << " - " << result[i] << endl;
	cin.get();

	return 0;

	// second check the speed of a k-d tree search optimization of the algorithm on the cpu
	for (int i = 0; i < 50; i++)
	{
		init = clock();
		KNearestNeighborsCPUTreeSearch(dataArray, result, cnt);
		end = clock();
		timt += end - init;
	}
	cout << "[+] The optimived version of the algorithm on the CPU takes " << timt / 50 << " milliseconds" << endl;
	timt = 0;

	// third check the simple implementation speed on the gpu
	for (int i = 0; i < 50; i++)
	{
		init = clock();
		KNearestNeighborsGPU(dataArray, result, cnt);
		end = clock();
		timt += end - init;
	}
	cout << "[+] The non optimived version of the algorithm on the GPU takes " << timt / 50 << " milliseconds" << endl;
	timt = 0;

	// fourth check the time the k-d version takes to run on the gpu
	for (int i = 0; i < 50; i++)
	{
		init = clock();
		KNearestNeighborsGPUTreeSearch(dataArray, result, cnt);
		end = clock();
		timt += end - init;
	}
	cout << "[+] The optimived version of the algorithm on the GPU takes " << timt / 50 << " milliseconds" << endl;

	return 0;
}

// non optimized cpu algorithm
void KNearestNeighborsCPU(float3 *dataArray, int *result, int cnt)
{
	for (int i = 0; i < cnt; i++)
	{
		float minimumDist = 3.4028234664e38f, distance = 0;
		for (int j = 0; j < cnt; j++)
		{
			if (i != j)
			{
				distance = (dataArray[i].x - dataArray[j].x) * (dataArray[i].x - dataArray[j].x);
				distance += (dataArray[i].y - dataArray[j].y) * (dataArray[i].y - dataArray[j].y);
				distance += (dataArray[i].z - dataArray[j].z) * (dataArray[i].z - dataArray[j].z);

				if (distance < minimumDist)
				{
					minimumDist = distance;
					result[i] = j;
				}
			}
		}
	}
}
// optimized cpu algorithm
void KNearestNeighborsCPUTreeSearch(float3 *dataArray, int *result, int cnt)
{

}
// non optimized gpu algorithm
/*__global__ void KNearestNeighborsGPU(float3 *dataArray, int *result, int cnt)
{

}
// optimized gpu algorithm
__global__ void KNearestNeighborsGPUTreeSearch(float3 *dataArray, int *result, int cnt)
{

}
*/