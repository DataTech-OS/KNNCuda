#include "hip/hip_runtime.h"
#include <Windows.h>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

using namespace std;

void KNearestNeighborsCPU(float3 *dataArray, int *result, int cnt);
void KNearestNeighborsCPUTreeSearch(float3 *dataArray, int *result, int cnt);
__global__ void KNearestNeighborsGPU(float3 *dataArray, int *result, int cnt);
__global__ void KNearestNeighborsGPUTreeSearch(float3 *dataArray, int *result, int cnt);

int cnt = 10000;

int main(int argc, char **argv)
{
	clock_t init, end;

	// generate the data
	srand(time(NULL));

	int timt = 0;
	float3 *dataArray = new float3[cnt];
	int *result = new int[cnt];

	for (int i = 0; i < cnt; i++)
	{
		dataArray[i].x = (rand() / 10000) - 5000;
		dataArray[i].y = (rand() / 10000) - 5000;
		dataArray[i].z = (rand() / 10000) - 5000;
	}
	// first check the speed of the algorithm takes on the cpu
	for (int i = 0; i < 10; i++)
	{
		init = clock();
		KNearestNeighborsCPU(dataArray, result, cnt);
		end = clock();
		timt += end - init;

		cout << "Iteration number " << i << " took " << end - init << " milliseconds" << endl;
	}
	cout << "[+] The non optimized version of the algorithm on the CPU takes " << timt / 10 << " milliseconds" << endl;
	timt = 0;

	for (int i = 0; i < 10; i++)
		cout << i << " - " << result[i] << endl;

	memset(result, 0, cnt);
	goto n;

	// second check the speed of a k-d tree search optimization of the algorithm on the cpu
	for (int i = 0; i < 10; i++)
	{
		init = clock();
		KNearestNeighborsCPUTreeSearch(dataArray, result, cnt);
		end = clock();
		timt += end - init;
	}
	cout << "[+] The optimized version of the algorithm on the CPU takes " << timt / 10 << " milliseconds" << endl;
	timt = 0;

n:
	// allocate and copy memory to the gpu
	float3 *deviceData;
	int *deviceResult;

	if (hipMalloc(&deviceData, sizeof(float3)*cnt) != hipSuccess)
	{
		cout << "[+] Unable to allocate GPU memory" << endl;
		return -1;
	}
	if (hipMalloc(&deviceResult, sizeof(int)*cnt) != hipSuccess)
	{
		cout << "[+] Unable to allocate GPU memory" << endl;
		return -2;
	}
	if (hipMemcpy(deviceData, dataArray, sizeof(float3)*cnt, hipMemcpyHostToDevice) != hipSuccess)
	{
		cout << "[+] Error in moving memory to the GPU" << endl;
		return -3;
	}

	// third check the simple implementation speed on the gpu
	for (int i = 0; i < 10; i++)
	{
		init = clock();
		KNearestNeighborsGPU<<< (cnt / 128) + 1, 128 >>>(deviceData, deviceResult, cnt);
		hipMemcpy(result, deviceResult, sizeof(int)*cnt, hipMemcpyDeviceToHost);
		end = clock();
		timt += end - init;

		cout << "Iteration number " << i << " took " << end - init << " milliseconds" << endl;
	}
	cout << "[+] The non optimized version of the algorithm on the GPU takes " << timt / 10 << " milliseconds" << endl;
	timt = 0;

	for (int i = 0; i < 10; i++)
		cout << i << " - " << result[i] << endl;
	cin.get();

	return 0;

	// fourth check the time the k-d version takes to run on the gpu
	for (int i = 0; i < 50; i++)
	{
		init = clock();
		//KNearestNeighborsGPUTreeSearch(dataArray, result, cnt);
		end = clock();
		timt += end - init;
	}
	cout << "[+] The optimized version of the algorithm on the GPU takes " << timt / 50 << " milliseconds" << endl;

	return 0;
}

// non optimized cpu algorithm
void KNearestNeighborsCPU(float3 *dataArray, int *result, int cnt)
{
	for (int i = 0; i < cnt; i++)
	{
		float minimumDist = 3.4028234664e38f, distance = 0;
		for (int j = 0; j < cnt; j++)
		{
			if (i != j)
			{
				distance = (dataArray[i].x - dataArray[j].x) * (dataArray[i].x - dataArray[j].x);
				distance += (dataArray[i].y - dataArray[j].y) * (dataArray[i].y - dataArray[j].y);
				distance += (dataArray[i].z - dataArray[j].z) * (dataArray[i].z - dataArray[j].z);

				if (distance < minimumDist)
				{
					minimumDist = distance;
					result[i] = j;
				}
			}
		}
	}
}

// optimized cpu algorithm
void KNearestNeighborsCPUTreeSearch(float3 *dataArray, int *result, int cnt)
{

}

// non optimized gpu algorithm
__global__ void KNearestNeighborsGPU(float3 *dataArray, int *result, int cnt)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < cnt)
	{
		float3 point = dataArray[id];
		float minimumDist = 3.4028234664e38f, distance = 0;

		for (int j = 0; j < cnt; j++)
		{
			if (id != j)
			{
				float3 current = dataArray[j];
				distance = (point.x - current.x) * (point.x - current.x);
				distance += (point.y - current.y) * (point.y - current.y);
				distance += (point.z - current.z) * (point.z - current.z);
				
				if (distance < minimumDist)
				{
					minimumDist = distance;
					result[id] = j;
				}
			}
		}
	}
}

// optimized gpu algorithm
__global__ void KNearestNeighborsGPUTreeSearch(float3 *dataArray, int *result, int cnt)
{

}